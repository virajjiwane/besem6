#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<math.h>
#include<iostream>
using namespace std;

__global__ void sum(float* input)
{
int tid = threadIdx.x;
int step_size = 1;
int number_of_threads = blockDim.x;
float aux_size = (float)number_of_threads;

while (number_of_threads > 0)
{
if (tid < number_of_threads)
{
int fst = tid*step_size * 2;
int snd = fst + step_size;
/*if (input[fst] > input[snd] && input[snd] > 0)
input[fst] = input[snd];*/
input[fst] = input[fst] + input[snd];

}
step_size = step_size*2;
if (number_of_threads != 1)
{
aux_size = aux_size / 2;
number_of_threads = (int)ceil(aux_size);
}
else
number_of_threads = 0;
}
}

int main(int argc, char const* argv[])
{
int count = 10;
float size = count * sizeof(float);
float  h[10];
//srand(100);
cout << "AARAY : " <<endl;
for (int i = 0; i < count; i++)
{
cout << i << "th element" << endl;
cin >> h[i]; //rand() % count;

}
cout << "AARAY : " << endl;
for (int i = 0; i < count; i++)
{
cout << h[i] << " ";
}

float* d;
hipMalloc(&d, size);
hipMemcpy(d, h, size, hipMemcpyHostToDevice);
sum << <1, (count / 2) + 1 >> > (d);
float result;
hipMemcpy(&result, d, sizeof(float), hipMemcpyDeviceToHost);
cout << "SUM :" << (float)result/(float)count << endl;
float mean = (float)result / (float)count;
for (int i = 0; i < count; i++)
{
h[i] = (h[i] - mean) * (h[i] - mean);
cout << h[i] << " ";
}
float* f;
hipMalloc(&f, size);
hipMemcpy(f, h, size, hipMemcpyHostToDevice);
sum << <1, (count / 2) + 1 >> > (f);
float resultfil;
hipMemcpy(&resultfil, f, sizeof(float), hipMemcpyDeviceToHost);
cout << "SD :" <<sqrt(resultfil)<< endl;

getchar();
hipFree(d);
hipFree(f);
return 0;
}
